#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int width, int maxIteration, int* result) {
	// To avoid error caused by the floating number, use the following pseudo code
	//
	// float x = lowerX + thisX * stepX;
	// float y = lowerY + thisY * stepY;
	int thisX = blockIdx.x * blockDim.x + threadIdx.x;
	int thisY = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_index = thisY * width + thisX;

	float c_re = lowerX + thisX * stepX;
	float c_im = lowerY + thisY * stepY;

	float z_re = c_re, z_im = c_im;
	int i;
	for(i = 0; i < maxIteration; ++i)
	{
		if(z_re * z_re + z_im * z_im > 4.f)
			break;

		float new_re = z_re * z_re - z_im * z_im;
		float new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;
	}

	result[thread_index] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
	//resX : width(1600) , resY : height(1200)
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

	hipHostRegister(img, resX * resY * sizeof(int), hipHostRegisterMapped);

	int* dev_img;
	//last arg : flag(must be 0 for now -> cuda_documentation said)
	hipHostGetDevicePointer((void **) &dev_img, (void *) img, 0);

	{
		dim3 dimBlock(32, 6);
		dim3 dimGrid(resX / 32, resY / 6);
		mandelKernel<<<dimGrid, dimBlock>>>(lowerX, lowerY, stepX, stepY, resX, maxIterations, dev_img);
		//I think can remove it? synchronize looks like don't have any impact on result or performance
		hipDeviceSynchronize();
	}

	hipHostUnregister(img);
	return;
}
